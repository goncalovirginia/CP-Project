#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "histogram_eq.h"

namespace cp {

	constexpr int HISTOGRAM_LENGTH = 256;
	constexpr int THREADS_PER_BLOCK = 2048;
	constexpr int TILE_WIDTH = 64;

	static float inline prob(const int x, const int size) {
		return static_cast<float>(x) / static_cast<float>(size);
	}

	__device__ static unsigned char inline gpu_clamp(unsigned char x) {
		return min(max(x, static_cast<unsigned char>(0)), static_cast<unsigned char>(255));
	}

	__device__ static unsigned char inline gpu_correct_color(float cdf_val, float cdf_min) {
		return gpu_clamp(static_cast<unsigned char>(255 * (cdf_val - cdf_min) / (1 - cdf_min)));
	}

	__global__ void correct_color(unsigned char *uchar_image, float *cdf, float cdf_min, int size_channels) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < size_channels) {
			uchar_image[idx] = gpu_correct_color(cdf[uchar_image[idx]], cdf_min);
		}
	}

	__global__ void percentageTo255(const float *input, unsigned char *output, int size) {
		int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx < size) {
			output[idx] = static_cast<unsigned char>(255 * input[idx]);
		}
	}

	__global__ void grayScale(const unsigned char *rgb_image, unsigned char *gray_image, int width, int height) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x < width && y < height) {
			int idx = y * width + x;
			unsigned char r = rgb_image[idx * 3];
			unsigned char g = rgb_image[idx * 3 + 1];
			unsigned char b = rgb_image[idx * 3 + 2];
			gray_image[idx] = static_cast<unsigned char>(0.21 * r + 0.71 * g + 0.07 * b);
		}
	}

	__global__ void computeHistogram(const unsigned char *gray_image, int *local_histograms, int width, int height) {
		extern __shared__ int local_hist[];

		if (threadIdx.x < HISTOGRAM_LENGTH) {
			local_hist[threadIdx.x] = 0;
		}
		__syncthreads();

		int i = blockIdx.x * blockDim.x + threadIdx.x;

		while (i < width * height) {
			atomicAdd(&local_hist[gray_image[i]], 1);
			i += blockDim.x * gridDim.x;
		}
		__syncthreads();

		if (threadIdx.x < HISTOGRAM_LENGTH) {
			atomicAdd(&local_histograms[blockIdx.x * HISTOGRAM_LENGTH + threadIdx.x], local_hist[threadIdx.x]);
		}
	}

	__global__ void mergeHistograms(int *local_histograms, int *global_histogram, int num_blocks) {
		int idx = threadIdx.x;
		int sum = 0;

		for (int i = 0; i < num_blocks; i++) {
			sum += local_histograms[i * HISTOGRAM_LENGTH + idx];
		}
		global_histogram[idx] = sum;
	}

	static float computeCDF(float (&cdf)[HISTOGRAM_LENGTH], int (&histogram)[HISTOGRAM_LENGTH], const int size) {
		#pragma omp parallel for schedule(static, 64)
		for (int i = 0; i < HISTOGRAM_LENGTH; ++i) {
			cdf[i] = prob(histogram[i], size);
		}

		float cdf_min = cdf[0];

		for (int i = 1; i < HISTOGRAM_LENGTH; ++i) {
			cdf[i] += cdf[i-1];
			cdf_min = std::min(cdf_min, cdf[i]);
		}

		return cdf_min;
	}

	__global__ void computeOutputImage(const unsigned char *uchar_image, float *output_image_data, int size_channels) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx < size_channels) {
			output_image_data[idx] = static_cast<float>(uchar_image[idx]) / 255.0f;
		}
	}

	static void histogram_equalization(const int width, const int height, const int size, const int size_channels,
									   int (&histogram)[HISTOGRAM_LENGTH], float (&cdf)[HISTOGRAM_LENGTH],
									   const float *gpu_input_image, unsigned char *gpu_uchar_image, unsigned char *gpu_rgb_image,
									   unsigned char *gpu_gray_image, int *gpu_local_histograms, int *gpu_global_histogram,
									   float *gpu_cdf, float *gpu_output_image_data) {

		percentageTo255<<<(size_channels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(gpu_input_image,gpu_uchar_image,size_channels);
		hipDeviceSynchronize();

		dim3 gridDim((width - 1) / TILE_WIDTH + 1, (height - 1) / TILE_WIDTH + 1);
		dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
		grayScale<<<gridDim, blockDim>>>(gpu_uchar_image, gpu_gray_image, width, height);
		hipDeviceSynchronize();

		int numBlocks = (width * height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		std::fill(histogram, histogram + HISTOGRAM_LENGTH, 0);
		hipMemset(gpu_local_histograms, 0, numBlocks * HISTOGRAM_LENGTH * sizeof(int));
		hipMemset(gpu_global_histogram, 0, HISTOGRAM_LENGTH * sizeof(int));
		computeHistogram<<<numBlocks, THREADS_PER_BLOCK, HISTOGRAM_LENGTH * sizeof(int)>>>(gpu_gray_image,gpu_local_histograms, width,height);
		hipDeviceSynchronize();
		mergeHistograms<<<1, HISTOGRAM_LENGTH>>>(gpu_local_histograms, gpu_global_histogram, numBlocks);
		hipDeviceSynchronize();
		hipMemcpy(histogram, gpu_global_histogram, HISTOGRAM_LENGTH * sizeof(int), hipMemcpyDeviceToHost);

		float cdf_min = computeCDF(cdf, histogram, size);

		hipMemcpy(gpu_cdf, cdf, HISTOGRAM_LENGTH * sizeof(float), hipMemcpyHostToDevice);
		correct_color<<<(size_channels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(gpu_uchar_image, gpu_cdf, cdf_min, size_channels);
		hipDeviceSynchronize();

		computeOutputImage<<<(size_channels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(gpu_uchar_image, gpu_output_image_data, size_channels);
		hipDeviceSynchronize();
	}

	wbImage_t iterative_histogram_equalization(wbImage_t &input_image, int iterations) {
		const auto width = wbImage_getWidth(input_image);
		const auto height = wbImage_getHeight(input_image);
		constexpr auto channels = 3;
		const auto size = width * height;
		const auto size_channels = size * channels;

		wbImage_t output_image = wbImage_new(width, height, channels);
		float *input_image_data = wbImage_getData(input_image);
		float *output_image_data = wbImage_getData(output_image);

		std::shared_ptr<unsigned char[]> uchar_image(new unsigned char[size_channels]);
		std::shared_ptr<unsigned char[]> gray_image(new unsigned char[size]);

		int histogram[HISTOGRAM_LENGTH];
		float cdf[HISTOGRAM_LENGTH];

		float *gpu_input, *gpu_cdf, *gpu_output_image_data;
		unsigned char *gpu_uchar_image, *gpu_rgb_image, *gpu_gray_image;
		int *gpu_local_histograms, *gpu_global_histogram;

		int numBlocks = (width * height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

		hipMalloc(&gpu_input, size_channels * sizeof(float));
		hipMalloc(&gpu_uchar_image, size_channels * sizeof(unsigned char));
		hipMalloc(&gpu_rgb_image, size_channels * sizeof(unsigned char));
		hipMalloc(&gpu_gray_image, size * sizeof(unsigned char));
		hipMalloc(&gpu_local_histograms, numBlocks * HISTOGRAM_LENGTH * sizeof(int));
		hipMalloc(&gpu_global_histogram, HISTOGRAM_LENGTH * sizeof(int));
		hipMalloc(&gpu_cdf, HISTOGRAM_LENGTH * sizeof(float));
		hipMalloc(&gpu_output_image_data, size_channels * sizeof(float));

		hipMemcpy(gpu_input, input_image_data, size_channels * sizeof(float), hipMemcpyHostToDevice);

		for (int i = 0; i < iterations; i++) {
			histogram_equalization(width, height, size, size_channels,histogram, cdf,
								   gpu_input, gpu_uchar_image, gpu_rgb_image, gpu_gray_image,
								   gpu_local_histograms, gpu_global_histogram,gpu_cdf, gpu_output_image_data);

			input_image_data = output_image_data;
		}

		hipMemcpy(output_image_data, gpu_output_image_data, size_channels * sizeof(float), hipMemcpyDeviceToHost);

		hipFree(gpu_input);
		hipFree(gpu_uchar_image);
		hipFree(gpu_rgb_image);
		hipFree(gpu_gray_image);
		hipFree(gpu_local_histograms);
		hipFree(gpu_global_histogram);
		hipFree(gpu_cdf);
		hipFree(gpu_output_image_data);

		return output_image;
	}
}